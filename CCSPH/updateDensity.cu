#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "stdio.h"
#include "smoothingKernels.cuh"

__global__ void updateDensity(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	double rhoRef = pparams->rRef;
	double cSquared = pparams->cSound * pparams->cSound;

	if (index < (*pparams).nTotal) {  //operate over all particles

		double dt = pparams->dt;

		//store the updated sorted density
		pparticles->sortedRho[index] += pparticles->sorteddRhodt[index] * dt;
		pparticles->sortedPressure[index] = computePressure(pparticles->sortedRho[index], rhoRef, cSquared);
		//printf("%d \n", pparticles->sortedRho[index]);

#if 0  //limit the density 
		if (> ) {
		}

#endif

	}

	return;
}