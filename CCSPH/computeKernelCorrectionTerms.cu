#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <stdio.h>
#include "smoothingKernels.cuh"

//__device__ double poly6(double cd, double r);
//__device__ double spikyImprovedD(double cd, double r);
//__device__ double computePressure(double rhoi, double rhoRef, double tenVMaxSq);

__device__ void computeGrads(int2 neighboor, double posX, double posY, struct paramsType* params, struct particleStructure* particles, double* changes);

__global__ void computeKernelCorrectionTerms(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	double det_threshold = 0.6;
	double rhoRef = pparams->rRef;
	double cSquared = pparams->cSound * pparams->cSound;

	//read primary particle data - this is sorted data
	double posXi = pparticles->sortedX[index];
	double posYi = pparticles->sortedY[index];

	int origIndex = pparticles->gridParticleIndex[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);
	int2 gridPos = { tempX,tempY };

	//examine neighbooring cells
	double changes[3] = { 0,0,0 };  //pointer to array of {fx,fy,XSPHx,XSPHy}

	for (int y = -1; y <= 1; y++) {
		int newY = gridPos.y + y;
		if ((newY > -1) && (newY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int newX = gridPos.x + x;
				if ((newX > -1) && (newX < (*pparams).nCellsX)) {
					int2 neighboor = { newX,newY };  //2D index in grid
					computeGrads(neighboor, posXi, posYi, pparams, pparticles, changes);

				}
			}
		}

	}


	double det = changes[0] * changes[2] - changes[1] * changes[1];

	pparticles->det_values[index] = det;

	if (det < det_threshold) {
		pparticles->sortedA11[index] = 1;
		pparticles->sortedA12[index] = 0;
		pparticles->sortedA22[index] = 1;
	}
	else {
		pparticles->sortedA11[index] = -changes[0];
		pparticles->sortedA12[index] = -changes[1];
		pparticles->sortedA22[index] = -changes[2];
	}

	//if (index == 2500) {
	//	printf("A11 for the particle with index %d is %f\n", index, pparticles->sortedA11[index]);
	//}


	//printf("%d\n",pparticles->sorteddRhodt[index]);

	//debug
#if (0)
	int orig = pparticles->gridParticleIndex[index];
	if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
		printf("AP; it=%u, P# %u; fx %f, fy %f, ", pparams->ind1, origIndex, pparticles->fx[originalIndex], pparticles->fy[originalIndex]);
	}
#endif


	return;
}
/*
__location__(global) void computePressure(particleStructure* pdParticles, paramsType* dParams)
{
	return __location__(global) void();
}
*/

// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double2 forcesInCell(int2 neighboor,int index,double posX,double posY,double velX, double velY, double rho,double2* posSorted,double2* velSorted,int* colorSorted, double2* dXSPHVelTemp,double2* massRadius,double* pRhoSorted,int* cellStart,int* cellEnd,const paramsType* params) {
__device__ void computeGrads(int2 neighboor, double posXi, double posYi, struct paramsType* pparams, struct particleStructure* pparticles, double* changes) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	double constWendlandD = pparams->constwendlandD;

	int startIndex = pparticles->cellStart[hash];
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];
		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {

			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double dx = (posXj - posXi);
			double dy = (posYj - posYi);
			double rSq = dx * dx + dy * dy;

			if ((rSq < 4 * pparams->h2) && (rSq > 0)) {
				double r = sqrt(rSq);
				double rOh = r / pparams->h;
				double Grad = (1 / r) * wendlandD(pparams->constwendlandD, rOh);

				double rhoj = pparticles->sortedRho[ind1];

				changes[0] += (dx * dx) * Grad * pparams->mass / rhoj;
				changes[1] += (dx * dy) * Grad * pparams->mass / rhoj;
				changes[2] += (dy * dy) * Grad * pparams->mass / rhoj;

			};
		};
	};

	return;
}