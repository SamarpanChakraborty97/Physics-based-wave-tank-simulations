#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"

__global__ void copySortedRhoToDensity(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	int origIndex = pparticles->gridParticleIndex[index];
	pparticles->density[origIndex] = pparticles->sortedRho[index];
	pparticles->pressure[origIndex] = pparticles->sortedPressure[index];

	//store density in an "unsorted form" the same order as x, y, vx, vy
	//at the begining of each step, new hash will be computed, and x, y, vx, vy density will be placed in their sorted arrays
	//therefore an array of unsorted denisty must exist




	return;
}