#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <stdio.h>
#include "smoothingKernels.cuh"

__global__ void computePressure(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//	if (colorSorted[index]==0) return;  //its a boundary particle
	double rhoRef = pparams->rRef;
	double cSquared = pparams->cSound * pparams->cSound;

	//read primary particle data - this is sorted data
	//double posXi = pparticles->sortedX[index];
	//double posYi = pparticles->sortedY[index];
	//double velXi = pparticles->sortedVx[index];
	//double velYi = pparticles->sortedVy[index];
	double rhoi = pparticles->sortedRho[index];
	double pressurei = computePressure(rhoi, rhoRef, cSquared);

	pparticles->sortedPressure[index] = pressurei;
	int originalIndex = pparticles->gridParticleIndex[index];
	pparticles->pressure[originalIndex] = pressurei;   //this one is used
}