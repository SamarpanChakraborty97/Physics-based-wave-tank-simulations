#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <stdio.h>


__global__ void calcHash(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < (pparams->nTotal)) {

		//compute grid position & clamp
		//remember cells go from [0,nCellsX-1] to give nCellsX # of cells
		int tempX = floor((pparticles->x[index] - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
		if (tempX >= (*pparams).nCellsX) { 
			tempX = (*pparams).nCellsX - 1; 
			printf("x-exceeded"); 
			
		}  //exceeded original x domain
		if (tempX < 0) { tempX = 0; 
			printf("x-negative"); 
		}  //lower than original x domain

		int tempY = floor((pparticles->y[index] - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);
		if (tempY >= (*pparams).nCellsY) { 
			tempY = (*pparams).nCellsY - 1; 
			printf("y-exceeded"); 
		}  //exceeded original y domain
		if (tempY < 0) { 
			tempY = 0; 
			printf("y-negative"); 
		}  //lower than original y domain

		//compute 1D hash value
		int hash = tempY * (*pparams).nCellsX + tempX;


		//won't happen
		if (hash < 0) { printf("hash<0 particle # %d\n", index); }

		if (hash >= (*pparams).nCellsTotal) { printf("hash>=max particle # %d\n", index); }  //this won't happen

		//store 
		pparticles->gridParticleHash[index] = hash;
		pparticles->gridParticleIndex[index] = index; //

		//dGridParticleIndex is just an array of consecutive indices right now, but it will be sorted later
	}



	return;
}