#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "stdio.h"

__global__ void updatePositionFreeParticles(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < (*pparams).nFree)
	{  //only operate over free particles

		double dt = pparams->dt;
		double x0 = pparams->relaxStart;
		double x1 = pparams->relaxEnd;
		double beta = pparams->betaDis;

		//int sortedIndex = pparticles->gridParticleIndex[index];
		//1st order Euler
#if 0
		pparticles->x[index] += pparticles->vx[index] * pparams->dt;
		pparticles->y[index] += pparticles->vy[index] * pparams->dt;
#endif


		//Leapfrog
#if 1
		//pparticles->sortedX[index] += pparticles->vxH[sortedIndex] * dt;
		//pparticles->sortedY[index] += pparticles->vyH[sortedIndex] * dt;

		pparticles->x[index] += pparticles->vxH[index] * dt;
		pparticles->y[index] += pparticles->vyH[index] * dt;

		if ((pparticles->x[index] >= x0) && (pparticles->x[index] <= x1)) {
			double f = 1 - dt * beta * ((pparticles->x[index] - x0) / (x1 - x0)) * ((pparticles->x[index] - x0) / (x1 - x0));
			pparticles->vxH[index] = pparticles->vxH[index] * f;
			pparticles->vyH[index] = pparticles->vyH[index] * f;
		}

		//pparticles->x[index] += pparticles->vx[index] * dt;
		//pparticles->y[index] += pparticles->vy[index] * dt;
#endif
		/*
		if ((pparticles->x[index] > relS) && (pparticles->x[index] < relE))
		{
			double dampV = b * dt * ((pparticles->x[index] - relS) / (relE - pparticles->x[index])) * ((pparticles->x[index] - relS) / (relE - pparticles->x[index]));
			pparticles->vx[index] = pparticles->vx[index] * (1 - dampV);
			pparticles->vy[index] = pparticles->vy[index] * (1 - dampV);
		}
		*/

		//debug
		if ((index == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
			printf("POS; it=%u, P# %u; px %f, py %f \n", pparams->ind1, index, pparticles->x[index], pparticles->y[index]);
		}



	} //end looping over free

	return;
}