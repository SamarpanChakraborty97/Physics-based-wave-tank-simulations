#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <stdio.h>
#include "smoothingKernels.cuh"

//__device__ double poly6(double cd, double r);
//__device__ double spikyImprovedD(double cd, double r);
//__device__ double computePressure(double rhoi, double rhoRef, double tenVMaxSq);

__device__ void forcesInCell2(int2 neighboor, int index, double posX, double posY, double velX, double velY, double rho, double pressurei, double a11, double a12, double a22, struct particleStructure* pparticles, struct paramsType* params, double* stateRates);
//__device__ void forcesInCell2(int2 neighboor, int index, double posX, double posY, double velX, double velY, double rho, double pressurei, struct particleStructure* pparticles, struct paramsType* params, double* stateRates);


__global__ void computedVdt(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//	if (colorSorted[index]==0) return;  //its a boundary particle
	double rhoRef = pparams->rRef;
	double cSquared = pparams->cSound * pparams->cSound;

	//read primary particle data - this is sorted data
	double posXi = pparticles->sortedX[index];
	double posYi = pparticles->sortedY[index];
	double velXi = pparticles->sortedVx[index];
	double velYi = pparticles->sortedVy[index];
	double rhoi = pparticles->sortedRho[index];
	//double rhoxi = pparticles->rhoGradX[index];
	//double rhoyi = pparticles->rhoGradY[index];
	double pressurei = computePressure(rhoi, rhoRef, cSquared);

	double a11I = pparticles->sortedA11[index];
	double a12I = pparticles->sortedA12[index];
	double a22I = pparticles->sortedA22[index];

	//debug
	int origIndex = pparticles->gridParticleIndex[index];
	if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
		printf("AP; it=%u, P# %u press. = %f, rhoi = %f, xi = %f, yi = %f \n", pparams->ind1, origIndex, pressurei, rhoi, posXi, posYi);
	}

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);
	int2 gridPos = { tempX,tempY };

	//examine neighbooring cells
	double stateRates[5] = { 0,0,0,0,0 };  //pointer to array of {fx,fy,XSPHx,XSPHy}

	for (int y = -1; y <= 1; y++) {
		int newY = gridPos.y + y;
		if ((newY > -1) && (newY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int newX = gridPos.x + x;
				if ((newX > -1) && (newX < (*pparams).nCellsX)) {
					int2 neighboor = { newX,newY };  //2D index in grid
					forcesInCell2(neighboor, index, posXi, posYi, velXi, velYi, rhoi, pressurei, a11I, a12I, a22I, pparticles, pparams, stateRates);
					//forcesInCell2(neighboor, index, posXi, posYi, velXi, velYi, rhoi, pressurei, pparticles, pparams, stateRates);
					//forcesTemp = forcesInCell2(neighboor,index,posXi,posYi,velXi,velYi,rhoi,pressurei,pparticles,pparams);
					//forces.x += forcesTemp.x;
					//forces.y += forcesTemp.y;
				}
			}
		}

	}

	/*
	//these are actually accelerations according to A. Vorobyev thesis
	//acceleration due to gravity is added at this point
	//write new forces back to original unsorted position
	int originalIndex = pparticles->gridParticleIndex[index];
	pparticles->fx[originalIndex] = forces.x;  //{x,y} //check sign
	pparticles->fy[originalIndex] = forces.y;  //{x,y}
	*/


	// revised - no longer requires copmute dRhoDt and SPHinfluence
	int originalIndex = pparticles->gridParticleIndex[index];
	pparticles->fx[originalIndex] = stateRates[0];
	pparticles->fy[originalIndex] = stateRates[1];

	pparticles->XSPHVelX[originalIndex] = stateRates[2];
	pparticles->XSPHVelY[originalIndex] = stateRates[3];

	pparticles->sorteddRhodt[index] = stateRates[4];
	//printf("%d\n",pparticles->sorteddRhodt[index]);

	//debug
#if (0)
	int orig = pparticles->gridParticleIndex[index];
	if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
		printf("AP; it=%u, P# %u; fx %f, fy %f, ", pparams->ind1, origIndex, pparticles->fx[originalIndex], pparticles->fy[originalIndex]);
	}
#endif


	return;
}
/*
__location__(global) void computePressure(particleStructure* pdParticles, paramsType* dParams)
{
	return __location__(global) void();
}
*/

// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double2 forcesInCell(int2 neighboor,int index,double posX,double posY,double velX, double velY, double rho,double2* posSorted,double2* velSorted,int* colorSorted, double2* dXSPHVelTemp,double2* massRadius,double* pRhoSorted,int* cellStart,int* cellEnd,const paramsType* params) {
__device__ void forcesInCell2(int2 neighboor, int index, double posXi, double posYi, double velXi, double velYi, double rhoi,  double pressurei, double a11I, double a12I, double a22I, struct particleStructure* pparticles, struct paramsType* pparams, double* stateRates) {
//__device__ void forcesInCell2(int2 neighboor, int index, double posXi, double posYi, double velXi, double velYi, double rhoi, double pressurei, struct particleStructure* pparticles, struct paramsType* pparams, double* stateRates) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	//required parameters
	double rhoRef = pparams->rRef;
	double cSquared = pparams->cSound * pparams->cSound;
	//double constantSpikyImprovedD = pparams->spikyImprovedD;
	//double constPoly6 = pparams->constDensity;
	double constWendland = pparams->constwendland;
	double constWendlandD = pparams->constwendlandD;

	int startIndex = pparticles->cellStart[hash];
	//double2 forces = {0,0};
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];
		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//remember to exclude self-force; stay within desired domain

			/*
			//DEBUG
			if (ind1==index) {
				printf("index == ind1, x position is %f  %f\n", pparticles->sortedX[ind1], posXi);
			}
			*/


			//if (ind1 != index) {

				//many SPH references cite pAB = pA - pB; where A is the primary particle.
			//following this convention we have
			//This gives a vector pointing from particle B to particle A

			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double dx = (posXi - posXj);
			double dy = (posYi - posYj);
			double rSq = dx * dx + dy * dy;

			if ((rSq <= 4*(*pparams).h2) && (rSq > 0)) {  //if they are close enough, proceede

				double h = pparams->h;
				double dist = sqrt(rSq);  //expensive but necessary
				double dvxij = velXi - pparticles->sortedVx[ind1];  //velocity of sending particle
				double dvyij = velYi - pparticles->sortedVy[ind1];
				double rOh = dist / h;
				double mj = pparams->mass;
				double rhoj = pparticles->sortedRho[ind1]; //rho of sender
				//double rhoxj = pparticles->rhoGradX[ind1];
				//double rhoyj = pparticles->rhoGradY[ind1];
				double pressurej = computePressure(rhoj, rhoRef, cSquared);
				//double Cij = sqrt(tenVMaxSq);

				double normalizedGradientInfluence = (1 / dist) * wendlandD(constWendlandD, rOh);
				/*

				double vMorTiInner = mj * (pparams->nu) * (rhoi + rhoj) / (rhoi * rhoj) * normalizedGradientInfluence;
				double vMorTiX = vMorTiInner * dvxij;
				double vMorTiY = vMorTiInner * dvyij;
				double dirVel = dvxij * dx + dvyij * dy;
				double rhoBarij = (rhoi + rhoj) / 2;      //used in XSPH as well

				double vTix = vMorTiX;  //it gets vTi regardless
				double vTiy = vMorTiY;

				if (dirVel < 0)   //it may get additional terms
				{
					//				double muij        = h*dirVel/(rSq+0.01*h*h);
					double muij = h  * dirVel / rSq ;

					double addedViscosity1 = mj * pparams->viscoBeta * muij * muij / rhoBarij * normalizedGradientInfluence;
					double addedViscosity2 = mj * (-pparams->viscoAlpha) * Cij* muij / rhoBarij * normalizedGradientInfluence;
					double addedViscosity = addedViscosity1 + addedViscosity2;
					vTix += addedViscosity * dx ;  //already has 1/|rij|
					vTiy += addedViscosity * dy;
				}

				double sharedTerm = mj * (pressurei / (rhoi * rhoi) + pressurej / (rhoj * rhoj)) * normalizedGradientInfluence;
				//double sharedTerm = mj * (pressurei / (rhoi * rhoi) + pressurej / (rhoj * rhoj)) * spikyImprovedD(constantSpikyImprovedD, rOh);

				double term1X = sharedTerm * dx;
				double term1Y = sharedTerm * dy;


				//these are actually accelerations
				stateRates[0] += -term1X ;  //fx
				stateRates[1] += -term1Y ;  //fy

				*/

				double vMorTiInner = mj * (pparams->nu) * (rhoi + rhoj) / (rhoi * rhoj) * normalizedGradientInfluence;
				double vTiX = vMorTiInner * dvxij;
				double vTiY = vMorTiInner * dvyij;

				double a11J = pparticles->sortedA11[ind1];
				double a12J = pparticles->sortedA12[ind1];
				double a22J = pparticles->sortedA22[ind1];

				double A11 = 0.5 * (a11I + a11J);
				double A12 = 0.5 * (a12I + a12J);
				double A22 = 0.5 * (a22I + a22J);

				double det = A11 * A22 - A12 * A12;
				double B11 = (1 / det) * A22;
				double B12 = -(1 / det) * A12;
				double B22 = (1 / det) * A11;

				double dirVel = dvxij * dx + dvyij * dy;
				double rhoBarij = (rhoi + rhoj) / 2;      //used in XSPH as well
				double Cij = pparams->cSound;  //mean speed of sound

				double sharedTerm = (1/rhoi) * (mj/rhoj) * (pressurei + pressurej) * normalizedGradientInfluence;
				//double sharedTerm = mj * ((pressurei / (rhoi * rhoi)) + (pressurej / (rhoj * rhoj))) * normalizedGradientInfluence;

				//double term1X = sharedTerm * dx;
				//double term1Y = sharedTerm * dy;
				double term1X = sharedTerm * (B11 * dx + B12 * dy);
				double term1Y = sharedTerm * (B12 * dx + B22 * dy);


				if (dirVel < 0)   //it may get additional terms
				{
					//				double muij        = h*dirVel/(rSq+0.01*h*h);
					double muij = h * dirVel / rSq;

					double addedViscosity1 = mj * pparams->viscoBeta * muij * muij / rhoBarij * normalizedGradientInfluence;
					//double addedViscosity2 = pparams->rRef * (mj/rhoj) * (-pparams->viscoAlpha) * Cij* muij / rhoBarij * normalizedGradientInfluence;
					double addedViscosity2 = mj * (-pparams->viscoAlpha) * Cij * muij / rhoBarij * normalizedGradientInfluence;
					double addedViscosity = addedViscosity1 + addedViscosity2;
					//term1X += addedViscosity * dx;  //already has 1/|rij| 
					//term1Y += addedViscosity * dy;
					term1X += addedViscosity * (B11 * dx + B12 * dy);  //already has 1/|rij| 
					term1Y += addedViscosity * (B12 * dx + B22 * dy);
				}

				//these are actually accelerations
				//stateRates[0] += -term1X; //fx
				//stateRates[1] += -term1Y;  //fy

				stateRates[0] += -term1X + vTiX; //fx
				stateRates[1] += -term1Y + vTiY;  //fy

				//stateRates[0] += -term1X * (B11 * dx + B12 * dy) + vTiX; //fx
				//stateRates[1] += -term1Y * (B12 * dx + B22 * dy) + vTiY;  //fy

				//		forces.x += -term1X+vTix;
				//		forces.y += -term1Y+vTiy;


				//XSPH
				double mutualInfluence = pparams->epsilon * mj / rhoBarij * wendland(constWendland, rOh);
				stateRates[2] += -mutualInfluence * dvxij;  //XSPHx; dxvij = -dvxji; dvxji is called for in the definition
				stateRates[3] += -mutualInfluence * dvyij;  //XSPHy; dxvij = -dvxji; dvxji is called for in the definition

				//DRho/dt
				double term1 = mj * normalizedGradientInfluence * dirVel;
				//double term2 = (rhoj - rhoi) - 0.5 * (((rhoxi + rhoxj) * (-dx)) + ((rhoyi + rhoyj) * (-dy)));
				double term2 = (rhoj - rhoi);
				//double term2 = rho_grad[0] * (-dxij) + rho_grad[1] * (-dyij);
				double term3 = 2 * term2 * (mj / rhoj) * (((-dx * normalizedGradientInfluence * dx) + (-dy * normalizedGradientInfluence * dy)) / rSq);

				stateRates[4] += term1 + pparams->delta * h * pparams->cSound * term3;
				//stateRates[4] += mj * normalizedGradientInfluence * dirVel;
				//stateRates[4] += mj * spikyImprovedD(constantSpikyImprovedD, rOh) * (dvxij + dvyij);



				//debug
				int origIndex = pparticles->gridParticleIndex[index];
				if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
					int sendingIndex = pparticles->gridParticleIndex[ind1];
					printf("AP; it=%u, P# %u -> w/ %u, rhoj %f \n", pparams->ind1, origIndex, sendingIndex, rhoj);
				}



				/*
					int originalIndex = pparticles->gridParticleIndex[index];
					if (originalIndex==0)
					printf("host %d vtiy %10.10f term1Y %10.10f \n",originalIndex,vTiy, term1Y);
					*/
					/*
										int originalIndex = pparticles->gridParticleIndex[index];
										int originalInd1 = pparticles->gridParticleIndex[ind1];
										printf("host %d is close to %d %f %f %f \n",originalIndex, originalInd1,pressurei, dist,forces.y);
					*/

			}; //end excluding self
		}; //end looping over the cell
	};  //if start index is not empty

//return forces;
	return;
}