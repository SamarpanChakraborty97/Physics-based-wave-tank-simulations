#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "stdio.h"


__global__ void updateVelWithXSPH(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < (*pparams).nFree) {  //operate over free particles


	//DEBUG
		if ((index == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
			printf("V before XSPH; it=%u, P# %u; vx %f, vy %f \n", pparams->ind1, index, pparticles->vx[index], pparticles->vy[index]);
		}

		//correct the velcoity with the XSPH correction
		//pparticles->vx[index] += pparticles->XSPHVelX[index];
		//pparticles->vy[index] += pparticles->XSPHVelY[index];

		pparticles->vxH[index] += pparticles->XSPHVelX[index];
		pparticles->vyH[index] += pparticles->XSPHVelY[index];

		//DEBUG
		if ((index == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
			printf("V+XSPH; it=%u, P# %u; vx %f, vy %f \n", pparams->ind1, index, pparticles->vx[index], pparticles->vy[index]);
		}



#if 0  //limit the velocity
		if (> ) {
		}

#endif

	}


	return;
}