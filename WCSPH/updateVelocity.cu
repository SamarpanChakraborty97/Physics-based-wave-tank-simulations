#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <iostream>
#include <stdio.h>

__global__ void updateVelocity(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < (*pparams).nFree) {  //only operate over free particles
		//the accelerations are stored; simply add gravity to the y-dir
		//and incorporate the XSPH terms

	//F = m a;
	//a = F/m

//Euler 1st order
#if 0
		double accelX = pparticles->fx[index]; //fx actually stores an acceleration; no need to divide by mass
		double accelY = pparticles->fy[index] + pparams->gravity; //need to add gravity

		double dt = pparams->dt;
		double vNewx = pparticles->vx[index] + accelX * dt;  //XSPH is incorporated previously
		double vNewy = pparticles->vy[index] + accelY * dt;  //
		//store the updated velocity
		pparticles->vx[index] = vNewx;
		pparticles->vy[index] = vNewy;
#endif

		//Leapfrog
#if 1
			//UPDATE THE VELOCITY WITH XSPH VELOCITY
		double dt = pparams->dt;
		if (pparams->ind1 == 0) {
			//pparticles->vxH[index] = pparticles->vx[index] + pparticles->fx[index] * dt / 2;
			//pparticles->vyH[index] = pparticles->vy[index] + (pparticles->fy[index] + pparams->gravity) * dt / 2;

			pparticles->vxH[index] += pparticles->fx[index] * dt / 2;
			pparticles->vyH[index] += (pparticles->fy[index] + pparams->gravity) * dt / 2;
			pparticles->vx[index] = pparticles->vxH[index] + pparticles->fx[index] * dt / 2;
			pparticles->vy[index] = pparticles->vyH[index] + (pparticles->fy[index] + pparams->gravity) * dt / 2;
			//	printf("tStep==0\n");
			//	printf("vy of particle 1 %f\n",pparticles->vy[1]);
		}
		else {
			pparticles->vxH[index] += pparticles->fx[index] * dt;
			pparticles->vyH[index] += (pparticles->fy[index] + pparams->gravity) * dt;
			pparticles->vx[index] = pparticles->vxH[index] + pparticles->fx[index] * dt / 2;
			pparticles->vy[index] = pparticles->vyH[index] + (pparticles->fy[index] + pparams->gravity) * dt / 2;
		}
#endif








		//debug
		if ((index == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
			printf("V+update; it=%u, P# %u; vx %f, vy %f \n", pparams->ind1, pparams->DEBUGpNum, pparticles->vx[index], pparticles->vy[index]);
		}


	}
	return;
}