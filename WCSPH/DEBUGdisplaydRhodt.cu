#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH3DCPPCuda.h"
#include "stdio.h"

__global__ void DEBUGdisplaydRhodt(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < (*pparams).nTotal) {  //operate over all particles

	//DEBUG
		int origIndex = pparticles->gridParticleIndex[index];
		if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
			printf("dRho %f \n", pparticles->sorteddRhodt[index]);
		}


	}
	return;
}