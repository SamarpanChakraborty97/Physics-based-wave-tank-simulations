#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <iostream>
#include "smoothingKernels.cuh"
#include <stdio.h>

__device__ void rhoInCell2(int2 neighboor, int index, double posX, double posY, struct particleStructure* pparticles, struct paramsType* pparams, double* rhoRe);

__global__ void rhoRenormalize(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//read particle data - host particle
	double posXi = pparticles->sortedX[index];  //these are sorted, I is the receiver
	double posYi = pparticles->sortedY[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double rhoRe[2] = { 0, 0 };	//shiftingCoefficient calculation

	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					rhoInCell2(neighboor, index, posXi, posYi, pparticles, pparams, rhoRe);
				}
			}
		}
	}

	pparticles->rhoGradX[index] = rhoRe[0];
	pparticles->rhoGradY[index] = rhoRe[1];
	//pparticles->sortedShiftGradY = shiftGrad[1];

	//int originalIndex = pparticles->gridParticleIndex[index];


	return;
}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ void rhoInCell2(int2 neighboor, int index, double posXi, double posYi, struct particleStructure* pparticles, struct paramsType* pparams, double* rhoRe) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	double rhoi = pparticles->sortedRho[index];
	//double constantSpikyImprovedD = pparams->spikyImprovedD;
	double l1 = pparticles->L1[index];
	double l2 = pparticles->L2[index];
	double l3 = pparticles->L3[index];
	double l4 = pparticles->L4[index];
	double constWendlandD = pparams->constwendlandD;

	int startIndex = pparticles->cellStart[hash];

	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//remember to include self density
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			double rhoJ = pparticles->sortedRho[ind1]; // density of the neighbouring particle
			//double Cj = pparticles->sortedShift[ind1]; //shifrting coefficient of neighbouring particle
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dx = posXi - posXj;
			double dy = posYi - posYj;
			double rSq = dx * dx + dy * dy;
			//double diffSq = (*pparams).h2 - rSq;
			if ((rSq <= 4*(*pparams).h2) && (rSq > 0)) {
				double r = sqrt(rSq);
				double rOh = r / pparams->h;
				double normalizedGradientInfluence = (1 / r) * wendlandD(constWendlandD, rOh);
				//double shiftGradient = (Cj - Ci) * (m2 / rhoJ) * normalizedGradientInfluence;
				rhoRe[0] += (rhoJ - rhoi) * (m2 / rhoJ) * ((l1 * normalizedGradientInfluence * dx) + (l2 * normalizedGradientInfluence * dy));
				rhoRe[1] += (rhoJ - rhoi) * (m2 / rhoJ) * ((l3 * normalizedGradientInfluence * dx) + (l4 * normalizedGradientInfluence * dy));

				//debug
				int index = blockIdx.x * blockDim.x + threadIdx.x;
				int origIndex = pparticles->gridParticleIndex[index];
				if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
					int senderIdx = pparticles->gridParticleIndex[ind1];
					printf("Ini D; it=%u, P# %u -> w/ %u at %f and %f \n", pparams->ind1, pparams->DEBUGpNum, senderIdx, posXj, posYj);
				}


			}; //end checking closeness
		};  //end the for loop
	};//end the if statement

	return;
}