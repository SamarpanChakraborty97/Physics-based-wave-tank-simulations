#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <iostream>
#include "smoothingKernels.cuh"
#include <stdio.h>

__device__ void shiftGradInCell2(int2 neighboor, int index, double posX, double posY, double C, struct particleStructure* pparticles, struct paramsType* pparams, double* shiftGrad);

__global__ void computeShiftGrad(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nFree) return;

	//read particle data - host particle
	double posXi = pparticles->sortedX[index];  //these are sorted, I is the receiver
	double posYi = pparticles->sortedY[index];
	double Ci = pparticles->sortedShift[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double shiftGrad[2] = { 0, 0 };	//shiftingCoefficient calculation

	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					shiftGradInCell2(neighboor, index, posXi, posYi, Ci, pparticles, pparams, shiftGrad);
				}
			}
		}
	}

	//pparticles->sortedShiftGradX = shiftGrad[0];
	//pparticles->sortedShiftGradY = shiftGrad[1];

	int originalIndex = pparticles->gridParticleIndex[index];
	pparticles->shiftGradX[originalIndex] = shiftGrad[0];
	pparticles->shiftGradY[originalIndex] = shiftGrad[1];


	return;
}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ void shiftGradInCell2(int2 neighboor, int index, double posXi, double posYi, double Ci, struct particleStructure* pparticles, struct paramsType* pparams, double* shiftGrad) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;
	//double constantSpikyImprovedD = pparams->spikyImprovedD;
	double constWendlandD = pparams->constwendlandD;

	int startIndex = pparticles->cellStart[hash];

	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//remember to include self density
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			double rhoJ = pparticles->sortedRho[ind1]; // density of the neighbouring particle
			double Cj = pparticles->sortedShift[ind1]; //shifrting coefficient of neighbouring particle
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dx = posXi - posXj;
			double dy = posYi - posYj;
			double rSq = dx * dx + dy * dy;
			//double diffSq = (*pparams).h2 - rSq;
			if ((rSq <= 4*(*pparams).h2) && (rSq > 0)) {
				double r = sqrt(rSq);
				double rOh = r / pparams->h;
				double normalizedGradientInfluence = (1 / r) * wendlandD(constWendlandD, rOh);
				double shiftGradient = (Cj - Ci) * (m2 / rhoJ) * normalizedGradientInfluence;
				shiftGrad[0] += shiftGradient * dx;
				shiftGrad[1] += shiftGradient * dy;
				//debug
				int index = blockIdx.x * blockDim.x + threadIdx.x;
				int origIndex = pparticles->gridParticleIndex[index];
				if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
					int senderIdx = pparticles->gridParticleIndex[ind1];
					printf("Ini D; it=%u, P# %u -> w/ %u at %f and %f \n", pparams->ind1, pparams->DEBUGpNum, senderIdx, posXj, posYj);
				}


			}; //end checking closeness
		};  //end the for loop
	};//end the if statement

	return;
}