#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "stdio.h"

__global__ void updateShift(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < (*pparams).nFree)
	{  //only operate over free particles

		double dt = pparams->dt;
		//double b = pparams->beta;
		//double relS = pparams->relaxStart;
		//double relE = pparams->relaxEnd;

		double Afst = 1.5;
		double Afsm = 2;
		double A = 2;
		double velX = pparticles->vx[index];
		double velY = pparticles->vy[index];
		double velMag = sqrt(velX * velX + velY * velY);

		if (pparticles->posDiv[index] - Afst < 0) {
			double Afsc = (pparticles->posDiv[index] - Afst) / (Afsm - Afst);
			double drX = -Afsc * A * pparams->h * velMag * dt * pparticles->shiftGradX[index];
			double drY = -Afsc * A * pparams->h * velMag * dt * pparticles->shiftGradY[index];
			pparticles->x[index] += pparticles->vxH[index] * dt + drX;
			pparticles->y[index] += pparticles->vyH[index] * dt + drY;
		}
		else if (pparticles->posDiv[index] - Afst == 0) {
			double drX = -A * pparams->h * velMag * dt * pparticles->shiftGradX[index];
			double drY = -A * pparams->h * velMag * dt * pparticles->shiftGradY[index];
			pparticles->x[index] += pparticles->vxH[index] * dt + drX;
			pparticles->y[index] += pparticles->vyH[index] * dt + drY;
		}
		else {
			pparticles->x[index] += pparticles->vxH[index] * dt;
			pparticles->y[index] += pparticles->vyH[index] * dt;
		}
	}

	return;
}