#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <iostream>
#include <stdio.h>

__global__ void initializeDensity2(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	double density = 1000;
	pparticles->sortedRho[index] = density;   //this one is not used

	//	printf("sorted Rho of a particle %u is %f\n",index, density);
	int originalIndex = pparticles->gridParticleIndex[index];
	pparticles->density[originalIndex] = density;   //this one is used

	pparticles->sorteddRhodt[index] = 0;

	return;
}