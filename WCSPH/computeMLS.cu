#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <iostream>
#include "smoothingKernels.cuh"
#include <stdio.h>

__device__ void mlsInCell2(int2 neighboor, int index, double posX, double posY, struct particleStructure* pparticles, struct paramsType* pparams, double* L);

__global__ void computeMLS(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//read particle data - host particle
	double posXi = pparticles->sortedX[index];  //these are sorted, I is the receiver
	double posYi = pparticles->sortedY[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double L[4] = { 0, 0, 0, 0 };	//shiftingCoefficient calculation

	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					mlsInCell2(neighboor, index, posXi, posYi, pparticles, pparams, L);
				}
			}
		}
	}

	//pparticles->sortedShiftGradX = shiftGrad[0];
	//pparticles->sortedShiftGradY = shiftGrad[1];

	//int originalIndex = pparticles->gridParticleIndex[index];
	double det = (L[0] * L[3]) - (L[1] * L[2]);
	//printf("%d\n",det);
	//printf("%d\n", L[0]);
	//pparticles->L1[index] = L[3] / det;
	if (det == 0) {
		pparticles->L1[index] = 0;
		pparticles->L2[index] = 0;
		pparticles->L3[index] = 0;
		pparticles->L4[index] = 0;
	}
	else {
		pparticles->L1[index] = L[3] / det;
		pparticles->L2[index] = -L[1] / det;
		pparticles->L3[index] = -L[2] / det;
		pparticles->L4[index] = L[0] / det;
	}
	return;
}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ void mlsInCell2(int2 neighboor, int index, double posXi, double posYi, struct particleStructure* pparticles, struct paramsType* pparams, double* L) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;
	//double constantSpikyImprovedD = pparams->spikyImprovedD;
	double constWendlandD = pparams->constwendlandD;

	int startIndex = pparticles->cellStart[hash];

	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//remember to include self density
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			double rhoJ = pparticles->sortedRho[ind1]; // density of the neighbouring particle
			//double Cj = pparticles->sortedShift[ind1]; //shifrting coefficient of neighbouring particle
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dx = posXi - posXj;
			double dy = posYi - posYj;
			double rSq = dx * dx + dy * dy;
			//double diffSq = (*pparams).h2 - rSq;
			if ((rSq <= 4*(*pparams).h2) && (rSq > 0)) {
				double r = sqrt(rSq);
				double rOh = r / pparams->h;
				double normalizedGradientInfluence = (1 / r) * wendlandD(constWendlandD, rOh);
				//double shiftGradient = (Cj - Ci) * (m2 / rhoJ) * normalizedGradientInfluence;
				L[0] += (m2 / rhoJ) * (-dx * normalizedGradientInfluence * dx);
				L[1] += (m2 / rhoJ) * (-dx * normalizedGradientInfluence * dy);
				L[2] += (m2 / rhoJ) * (-dy * normalizedGradientInfluence * dx);
				L[3] += (m2 / rhoJ) * (-dy * normalizedGradientInfluence * dy);
				//debug
				int index = blockIdx.x * blockDim.x + threadIdx.x;
				int origIndex = pparticles->gridParticleIndex[index];
				if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
					int senderIdx = pparticles->gridParticleIndex[ind1];
					printf("Ini D; it=%u, P# %u -> w/ %u at %f and %f \n", pparams->ind1, pparams->DEBUGpNum, senderIdx, posXj, posYj);
				}


			}; //end checking closeness
		};  //end the for loop
	};//end the if statement

	return;
}