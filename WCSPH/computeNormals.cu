#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <stdio.h>
#include "smoothingKernels.cuh"

__device__ void forcesInCell2(int2 neighboor, int index, double posX, double posY,  struct particleStructure* pparticles, struct paramsType* params, double* stateRates);

__global__ void computeNormals(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//	if (colorSorted[index]==0) return;  //its a boundary particle
	double posXi = pparticles->sortedX[index];
	double posYi = pparticles->sortedY[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);
	int2 gridPos = { tempX,tempY };

	double stateRates[2] = { 0,0 };

	for (int y = -1; y <= 1; y++) {
		int newY = gridPos.y + y;
		if ((newY > -1) && (newY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int newX = gridPos.x + x;
				if ((newX > -1) && (newX < (*pparams).nCellsX)) {
					int2 neighboor = { newX,newY };  //2D index in grid
					forcesInCell2(neighboor, index, posXi, posYi, pparticles, pparams, stateRates);
					//forcesTemp = forcesInCell2(neighboor,index,posXi,posYi,velXi,velYi,rhoi,pressurei,pparticles,pparams);
					//forces.x += forcesTemp.x;
					//forces.y += forcesTemp.y;
				}
			}
		}

	}
	// revised - no longer requires copmute dRhoDt and SPHinfluence
	//int originalIndex = pparticles->gridParticleIndex[index];
	pparticles->sortedNormalX[index] = stateRates[0];
	pparticles->sortedNormalY[index] = stateRates[1];

	return;
}

__device__ void forcesInCell2(int2 neighboor, int index, double posXi, double posYi, struct particleStructure* pparticles, struct paramsType* pparams, double* stateRates) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	//required parameters
	double constWendlandD = pparams->constwendlandD;

	int startIndex = pparticles->cellStart[hash];
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];
		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double dx = (posXi - posXj);
			double dy = (posYi - posYj);
			double rSq = dx * dx + dy * dy;
			if ((rSq <= 4 * (*pparams).h2) && (rSq > 0)) {  //if they are close enough, proceede
				double h = pparams->h;
				double dist = sqrt(rSq);  //expensive but necessary
				double rOh = dist / h;
				double mj = pparticles->mass[0];
				double rhoj = pparticles->sortedRho[ind1]; //rho of sender
				double normalizedGradientInfluence = (1 / dist) * wendlandD(constWendlandD, rOh);
				stateRates[0] += h * (mj / rhoj) * normalizedGradientInfluence * dx;
				stateRates[1] += h * (mj / rhoj) * normalizedGradientInfluence * dy;
			};
		};
	};
	return;
};
				
	