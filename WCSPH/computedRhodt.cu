#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "smoothingKernels.cuh"
#include "stdio.h"

__device__ double dRhoInCell(int2 neighboor, int index, double posX, double posY, double velxi, double velyi, struct particleStructure* pparticles, struct paramsType* pparams);

__device__ void NeighGrad(int2 neighbour, int ind1, double posXj, double posYj, double velxj, double velyj, struct particleStructure* pparticles, struct paramsType* pparams, double* NeighbourGrad);

__global__ void computedRhodt(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//read particle data - host particle
	double posXi = pparticles->sortedX[index];  //these are sorted
	double posYi = pparticles->sortedY[index];
	double velxi = pparticles->sortedVx[index];
	double velyi = pparticles->sortedVy[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double dRho = 0;	//need density
	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					dRho += dRhoInCell(neighboor, index, posXi, posYi, velxi, velyi, pparticles, pparams);
				}
			}
		}
	}

#if 0 //limit the denisty
	if (newDensity > 6.0 * (*pparams).rRef)
	{
		newDensity = 6.0 * (*pparams).rRef;
		//printf("corrected density");
	}
#endif

	//write drhodt to sorted position
	pparticles->sorteddRhodt[index] = dRho;



	return;
}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ double dRhoInCell(int2 neighboor, int index, double posXi, double posYi, double velxi, double velyi, struct particleStructure* pparticles, struct paramsType* pparams) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	double constwendlandD = pparams->constwendlandD;
	double h = pparams->h;
	double rhoi = pparticles->sortedRho[index];

	int startIndex = pparticles->cellStart[hash];
	double drdt = 0;
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		double L_inv[4] = { 0, 0, 0, 0 };
		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//no reason to include self in drhodt
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dxij = (posXi - posXj);
			double dyij = (posYi - posYj);

			double rSq = dxij * dxij + dyij * dyij;
			if ((rSq < 4 * (*pparams).h2) && (rSq > 0)) {
				double dist = sqrt(rSq);
				double rOh = dist / h;
				double normalizedGradInfluence = (1 / dist) * wendlandD(constwendlandD, rOh);
				double normX = normalizedGradInfluence * dxij;
				double normY = normalizedGradInfluence * dyij;
				L_inv[0] += -dxij * normX;
				L_inv[1] += -dxij * normY;
				L_inv[2] += -dyij * normX;
				L_inv[3] += -dyij * normY;
			}
		}

		//now we have to calculate the inverse of the above matrix or tensor
		double L[4] = { 0, 0, 0, 0 };
		double det = (L_inv[0] * L_inv[3]) - (L_inv[1] * L_inv[2]);
		L[0] = L_inv[3] / det;
		L[1] = -L_inv[1] / det;
		L[2] = -L_inv[2] / det;
		L[3] = L_inv[0] / det;

		//we have to use that inverse in the calculation of density gradient renormalization done below
		double rho_grad[2] = { 0,0 };
		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//no reason to include self in drhodt
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dxij = (posXi - posXj);
			double dyij = (posYi - posYj);

			double rSq = dxij * dxij + dyij * dyij;
			if ((rSq < 4 * (*pparams).h2) && (rSq > 0)) {
				double dist = sqrt(rSq);
				double rOh = dist / h;
				double normalizedGradInfluence = (1 / dist) * wendlandD(constwendlandD, rOh);
				double normX = normalizedGradInfluence * dxij;
				double normY = normalizedGradInfluence * dyij;
				double rhoj = pparticles->sortedRho[ind1];

				rho_grad[0] += (rhoj - rhoi) * (L[0] * normX + L[1] * normY);
				rho_grad[1] += (rhoj - rhoi) * (L[2] * normX + L[3] * normY);
			}
		}

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//no reason to include self in drhodt
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dxij = (posXi - posXj);
			double dyij = (posYi - posYj);

			double rSq = dxij * dxij + dyij * dyij;
			if ((rSq < 4 * (*pparams).h2) && (rSq > 0)) {

				double vxj = pparticles->sortedVx[ind1];
				double vyj = pparticles->sortedVy[ind1];

				double dvxij = velxi - vxj;
				double dvyij = velyi - vyj;

				double dist = sqrt(rSq);
				double rOh = dist / pparams->h;
				double normalizedGradInfluence = (1 / dist) * wendlandD(constwendlandD, rOh);
				double normX = normalizedGradInfluence * dxij;
				double normY = normalizedGradInfluence * dyij;
				double rhoj = pparticles->sortedRho[ind1];

				int tempXj = floor((posXj - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
				int tempYj = floor((posYj - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

				int2 gridPosj = { tempXj,tempYj }; // grid position of host particle

				double NeighbourGrad[2] = { 0,0 };
				for (int y = -1; y <= 1; y++) {
					int currentY = gridPosj.y + y;
					if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

						for (int x = -1; x <= 1; x++) {
							int currentX = gridPosj.x + x;
							if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
								int2 neighbour = { currentX,currentY };  //2D index in grid
								NeighGrad(neighbour, ind1, posXj, posYj, vxj, vyj, pparticles, pparams, NeighbourGrad);
							}
						}
					}
				}
				double term1 = rhoi * (m2 / rhoj) * (dvxij * normX + dvyij * normY);
				double term2 = (NeighbourGrad[0] + rho_grad[0]) * (-dxij) + (NeighbourGrad[1] + rho_grad[1]) * (-dyij);
				double term3 = ((-dxij * normX) + (-dyij * normY)) / rSq;
				drdt += term1 + 2 * pparams->delta * h * 10 * pparams->vf * (m2 / rhoj) * (rhoj - rhoi - 0.5 * term2) * term3;


			}; //end checking closeness				


		};  //end the for loop
	};//end checking for populated cells

	return drdt;
}

__device__ void NeighGrad(int2 neighbour, int ind1, double posXj, double posYj, double velxj, double velyj, struct particleStructure* pparticles, struct paramsType* pparams, double* NeighbourGrad) {
	int hash2 = neighbour.y * (*pparams).nCellsX + neighbour.x;

	double constwendlandD = pparams->constwendlandD;
	double h = pparams->h;
	double rhoj = pparticles->sortedRho[ind1];

	int startIndex = pparticles->cellStart[hash2];
	double drdt = 0;
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash2];

		double L_inv[4] = { 0, 0, 0, 0 };
		for (int ind2 = startIndex; ind2 < endIndex; ind2++) {
			//no reason to include self in drhodt
			double posXk = pparticles->sortedX[ind2];  //get position of sending particles
			double posYk = pparticles->sortedY[ind2];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dxjk = (posXj - posXk);
			double dyjk = (posYj - posYk);

			double rSq = dxjk * dxjk + dyjk * dyjk;
			if ((rSq < 4 * (*pparams).h2) && (rSq > 0)) {
				double dist = sqrt(rSq);
				double rOh = dist / h;
				double normalizedGradInfluence = (1 / dist) * wendlandD(constwendlandD, rOh);
				double normX = normalizedGradInfluence * dxjk;
				double normY = normalizedGradInfluence * dyjk;
				L_inv[0] += -dxjk * normX;
				L_inv[1] += -dxjk * normY;
				L_inv[2] += -dyjk * normX;
				L_inv[3] += -dyjk * normY;
			}
		}

		//now we have to calculate the inverse of the above matrix or tensor
		double L[4] = { 0, 0, 0, 0 };
		double det = (L_inv[0] * L_inv[3]) - (L_inv[1] * L_inv[2]);
		L[0] = L_inv[3] / det;
		L[1] = -L_inv[1] / det;
		L[2] = -L_inv[2] / det;
		L[3] = L_inv[0] / det;

		//we have to use that inverse in the calculation of density gradient renormalization done below
		//double rho_grad[2] = { 0,0 };
		for (int ind2 = startIndex; ind2 < endIndex; ind2++) {
			//no reason to include self in drhodt
			double posXk = pparticles->sortedX[ind2];  //get position of sending particles
			double posYk = pparticles->sortedY[ind2];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dxjk = (posXj - posXk);
			double dyjk = (posYj - posYk);

			double rSq = dxjk * dxjk + dyjk * dyjk;
			if ((rSq < 4 * (*pparams).h2) && (rSq > 0)) {
				double dist = sqrt(rSq);
				double rOh = dist / h;
				double normalizedGradInfluence = (1 / dist) * wendlandD(constwendlandD, rOh);
				double normX = normalizedGradInfluence * dxjk;
				double normY = normalizedGradInfluence * dyjk;
				double rhok = pparticles->sortedRho[ind2];

				NeighbourGrad[0] += (rhok - rhoj) * (L[0] * normX + L[1] * normY);
				NeighbourGrad[1] += (rhok - rhoj) * (L[2] * normX + L[3] * normY);
			}
		}
	}
}