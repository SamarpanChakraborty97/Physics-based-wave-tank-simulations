#include "hip/hip_runtime.h"
//the cuda particles example, included in the SDK, was used as a reference
//the algorithms presented here were developed for this particular implmentation


#include "stdafx.h"
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "SPH2DCPPCuda.h"
#include "exportSystem.h"
#include "moveDataToGPU.h"
#include <omp.h>
//#include <iostream>


//note: had to modify C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v6.5\bin\nvcc.profile to get open mp working
//added "/openmp" with quotes in INCLUDE statement


#include "thrust/sort.h"
#include "thrust/device_vector.h"

#include <time.h>
#include "SPH2DCUDA.h"

#define threadsPerBlock 128

#define timeComponents (0)
#define DEBUGING (0)

__global__ void moveConstrainedParticlesD(struct particleStructure* pdParticles, struct paramsType* dParams, kinematicsFunctionStructure* dKinematicsFunction);
__global__ void moveConstrainedParticlesD2(struct particleStructure* pdParticles, struct paramsType* dParams, kinematicsFunctionStructure* dKinematicsFunction);
__global__ void calcHash(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void reorderAndFindCellStart(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void initializeDensity(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void initializeDensity2(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void computedVdt(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void computePressure(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void updateVelocity(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void computeXSPHInfluence(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void updatePositionFreeParticles(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void computeShift(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void computeShiftGrad(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void computeDiv(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void updateShift(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void computedRhodt(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void reinitializeDensity(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void moveFilteredDensityToDensity(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void updateDensity(struct particleStructure* pdParticles, struct paramsType* dParams);
__global__ void copySortedRhoToDensity(struct particleStructure* pparticles, struct paramsType* pparams);
__global__ void updateVelWithXSPH(struct particleStructure* pdParticles, struct paramsType* pdParams);

//DEBUG
__global__ void DEBUGdisplayDensity(struct particleStructure* pdParticles, struct paramsType* pdParams);
__global__ void DEBUGdisplaydRhodt(struct particleStructure* pdParticles, struct paramsType* pdParams);
__global__ void DEBUGdisplayMoveDensity(struct particleStructure* pdParticles, struct paramsType* pdParams);


void SPH2DCuda(particleStructure* particles, paramsType* params, std::vector<kinematicsFunctionStructure>* kinematicsFunction, std::string outputDir)
{


#if DEBUGING
#define DEBUG(...) \
	do{\
	__VA_ARGS__;\
	} while(0)
#else
#define DEBUG(...)\
	do {;} while(0)
#endif


#if timeComponents
	hipEvent_t start;
	hipEvent_t stop;
	float cudaTimeTemp;
	const int nProcedures = 21;
	int nChars = 30;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cudaTime[nProcedures];
	char* procedureNames[nProcedures];
	for (int ind1 = 0; ind1 < nProcedures; ind1++) {
		cudaTime[ind1] = 0;  //initialize cumulative timers
		procedureNames[ind1] = (char*)malloc(sizeof(char) * nChars);
		procedureNames[ind1] = ".";
	}
	procedureNames[0] = "calcHash                     ";
	procedureNames[1] = "thrust::sort_by_key..........";
	procedureNames[2] = "reorderAndFindCellStart......";
	procedureNames[3] = "computedVdt                  ";
	procedureNames[4] = "updateVelWithXSPH............";
	procedureNames[5] = "updateVelocity               ";
	procedureNames[6] = "updateDensity................";
	procedureNames[7] = "reinitializeDensity          ";
	procedureNames[8] = "moveFilteredDensityToDensity.";
	procedureNames[9] = "copySortedRhoToDensity       ";
	procedureNames[10] = "updatePositionFreeParticles..";
	procedureNames[11] = "moveConstrainedParticlesD    ";
	procedureNames[12] = "hipMemcpy...................";
	procedureNames[13] = "exportSystem***OMP***        ";
#define TIME(processNumber,...) \
	do {\
	hipEventRecord(start,0); \
	__VA_ARGS__; \
	hipEventRecord(stop,0); \
	hipEventSynchronize(stop);	\
	hipEventElapsedTime(&cudaTimeTemp, start, stop); \
	cudaTime[processNumber] += cudaTimeTemp; \
		} while(0)
#else
#define TIME(processNumber,...) __VA_ARGS__;
#endif

	//heartbeat
	hipEvent_t tStart, tStop;
	hipEventCreate(&tStart);
	hipEventCreate(&tStop);
	hipEventRecord(tStart);
	float elapsedTime;
	std::cout << '\n' << omp_get_max_threads();

	omp_set_num_threads(2);  //one thread writes the files, one thread issues GPU commands, more threads could only cause problems


	int smemSize = (threadsPerBlock + 1) * sizeof(int);
	int nTotal = params->nTotal;  //number of particles
	std::cout << '\n' << params->nTotal;
	int numberBlocks = (nTotal % threadsPerBlock != 0) ? (nTotal / threadsPerBlock + 1) : (nTotal / threadsPerBlock); //if condt ? outTrue : outFalse

	//int nFree = params->nFree;//number of free particles;
	int numberBlocksFree = (params->nFree % threadsPerBlock != 0) ? (nTotal / threadsPerBlock + 1) : (nTotal / threadsPerBlock); //if condt ? outTrue : outFalse

	int nC = params->nConstrained;  //number of constrained particles
	int numberBlocksConstrained = (nC % threadsPerBlock != 0) ? (nC / threadsPerBlock + 1) : (nC / threadsPerBlock); //if condt ? outTrue : outFalse

	//int nM = params->nMeasured; //number of constrained particles
	//int numberBlocksMeasured = (nM % threadsPerBlock != 0) ? (nM / threadsPerBlock + 1) : (nM / threadsPerBlock); //if condt ? outTrue : outFalse

	int nT = params->nTime; //number of time steps
	int storageStride = params->storageStride;
	int nStrides = nT / storageStride; //integer division implies floor();
	//printf("%d", nStrides);


	//define the # forces acting on the free particles 
	//in standard simulation, boundary particles administer forces as free particles
	//in Leonard-Jones type forces from boundary particles are handeled differently
	params->nFreeParticleForceDomain = params->nTotal;


	//--determine bin size--
	//bin size in x and y dimensions
	//its the maximum of the smoothing radii
	double binSize = 2 * particles->radius[0];

	//store the bin size and ...
	params->cellSizeRecip = 1 / binSize;

	//--determine domain limits--
	//domain limits are taken as the maximum and minimums in x & y directions from the INITIAL data set
	//if the user would like to specify larger domain limits, they can input 
	//constrained particles at specified locations.

	double2 globalMin = { particles->x[0],particles->y[0] };
	double2 globalMax = { particles->x[0],particles->y[0] };

	for (int ind1 = 0; ind1 < nTotal; ind1++) {
		if (particles->x[ind1] > globalMax.x) { globalMax.x = particles->x[ind1]; };
		if (particles->x[ind1] < globalMin.x) { globalMin.x = particles->x[ind1]; };
		if (particles->y[ind1] > globalMax.y) { globalMax.y = particles->y[ind1]; };
		if (particles->y[ind1] < globalMin.y) { globalMin.y = particles->y[ind1]; };
	}
	params->globalOriginX = globalMin.x;
	params->globalOriginY = globalMin.y;

	double2 delta = { globalMax.x - globalMin.x,globalMax.y - globalMin.y };
	//params->nCellsY     = fmod(delta.y,binSize) !=0 ? (int) (delta.y/binSize+1) : (int) (delta.y/binSize); //if condt ? outTrue : outFalse
	//params->nCellsX     = fmod(delta.x,binSize) !=0 ? (int) (delta.x/binSize+1) : (int) (delta.x/binSize); //if condt ? outTrue : outFalse
	params->nCellsY = (int)(delta.y / binSize) + 1; //cast to int is like floor, if its on the upper bin edge, then an additional bin survives
	params->nCellsX = (int)(delta.x / binSize) + 1;
	//std::cout << '\n' << params->nCellsTotal;
	params->nCellsTotal = (params->nCellsX) * (params->nCellsY);
	std::cout << '\n' << params->nCellsTotal;
	params->nFunctions = (*kinematicsFunction).size(); //store the number of kinematics functions
	//need the +1 becuase a particle might be on the max bin edge


	//precompute some constants used in the kernels, derived in Mathematica, and in Bindell.  Muller in 2D
	//the kernels as used are:
	//density   - 4/(pi h^8)*(h^2-r^2)^3
	//pressure  - -30/(h^5*pi)*(h-r)^2
	//viscosity - 40/(h^5*pi)*(h-r)


	double h = particles->radius[0];  //make particles 2x large here and above; use 2x smoothing length; revert in viscosity muij
	double h2 = h * h;
	double h8 = h2 * h2 * h2 * h2;
	double const1 = 4.0 / (3.141592654 * h2);  //poly6 2D              - verified
	double constPressure = (30.0 / (3.1415926535 * h2 * h2 * h));
	double constViscosity = 40.0 / (3.1415926535 * h2 * h2 * h);
	double quartic = 1 / (7 * 3.141592654 * h2);      //Liu normalized to h=1;
	double quarticD = -60 / (7 * 3.141592654 * h2 * h);  //Liu derivative
	double spikyImprovedD = -12 / (3.141592654 * h * h * h);
	double constWend = 7 / (4 * 3.1415926535 * h2);
	double constWendD = -35 / (4 * 3.1415926535 * h2 * h);

	params->h = h;
	params->h2 = h2;
	params->h8 = h8;
	params->constwendland = constWend;
	params->constwendlandD = constWendD;
	params->tenVMaxSq = (15 * params->vf) * (15 * params->vf);
	params->ind1 = 0;
	params->quartic = quartic;
	params->quarticD = quarticD;
	params->spikyImprovedD = spikyImprovedD;
	params->constwendland = constWend;
	params->constwendlandD = constWendD;

	//DEBUG related
	params->DEBUGinfo = 0;  //display debug info
	params->DEBUGpNum = 7171;  //debug particle #

	//make device pointers
	struct particleStructure* pdParticles = 0;  //device particles; 
	struct particleStructure* pdParticlesHostMirror = new struct particleStructure;  //resides on host, contains device pointers
	struct paramsType* pdParams = 0;  //device parameters
	struct kinematicsFunctionStructure* pdKinematicsFunction = 0;  //device kinematics

		//allocate memory and transfer data to GPU

	moveDataToGPU(particles, params, kinematicsFunction, &pdParticles, &pdParams, &pdKinematicsFunction, &pdParticlesHostMirror);
	//first, get constrained particles to their proper location
	//and reset the device time

	//moveConstrainedParticlesD2 increments the GPU copy of ind1
	//DEBUG - Nothing has been sorted, so the first time is no problem
	moveConstrainedParticlesD << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams, pdKinematicsFunction);
	hipMemcpy(pdParams, params, sizeof(paramsType), hipMemcpyHostToDevice); //reset the time (ind1)

	//initialize density and pressure
	calcHash << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams);  //compute hash value
	thrust::sort_by_key(thrust::device_ptr<int>(pdParticlesHostMirror->gridParticleHash), thrust::device_ptr<int>(pdParticlesHostMirror->gridParticleHash + (*params).nTotal), thrust::device_ptr<int>(pdParticlesHostMirror->gridParticleIndex));  //sort
	hipMemset(pdParticlesHostMirror->cellStart, 0xffffffff, (*params).nCellsTotal * sizeof(int));
	hipDeviceSynchronize();
	reorderAndFindCellStart << <numberBlocks, threadsPerBlock, smemSize >> > (pdParticles, pdParams);
	if (params->importDensity == 0) {  //need to initialize density
		//initializeDensity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams);     //compute
		initializeDensity2 << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams);     //compute
		printf("\nNot importing density; initializing density\n");
	}
	else { printf("\nImporting density\n"); }

	computePressure << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams);     //compute pressure
	//hipMemcpy(particles->density,pdParticlesHostMirror->sortedRho,nTotal*sizeof(double),hipMemcpyDeviceToHost);  // no need
	//transfer memory back to host in anticipation of writing 0th file

	//enter loop
	unsigned int iteration = 0;  //global iteration #
	while (iteration < nT)
	{
		unsigned int snapshotIteration = iteration / storageStride;  //copy the current iteration to use in the exported file name
///////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////TRANSFER CURRENT DATA TO HOST IN ANTICIPATION OF FILE EXPORT//////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////
		hipDeviceSynchronize();
		TIME(10, hipMemcpy(particles->x, pdParticlesHostMirror->x, nTotal * sizeof(double), hipMemcpyDeviceToHost););
		TIME(10, hipMemcpy(particles->y, pdParticlesHostMirror->y, nTotal * sizeof(double), hipMemcpyDeviceToHost););
		//TIME(10, hipMemcpy(particles->vx, pdParticlesHostMirror->vx, nTotal * sizeof(double), hipMemcpyDeviceToHost););
		//TIME(10, hipMemcpy(particles->vy, pdParticlesHostMirror->vy, nTotal * sizeof(double), hipMemcpyDeviceToHost););
		TIME(10, hipMemcpy(particles->density, pdParticlesHostMirror->density, nTotal * sizeof(double), hipMemcpyDeviceToHost););
		//TIME(10, hipMemcpy(particles->pressure, pdParticlesHostMirror->pressure, nTotal * sizeof(double), hipMemcpyDeviceToHost););
		//std::cout <<'\n' <<  params->nMeasured;

#if timeComponents																		//
		hipEventRecord(stop, 0);												//
		hipEventSynchronize(stop);												//
#endif	

#pragma omp parallel sections
		{
#pragma omp section
			{
				/////////////////////////////////////////////////////////////////////////////////////////////////////					
								//EXPORT DATA from previous transfer while the next iterations are being computed
								//no cuda calls in this section
				/////////////////////////////////////////////////////////////////////////////////////////////////////
#if timeComponents
			//hipEventRecord(start[13],0); 
				double fileTimeStart = omp_get_wtime();
#endif
				int	output = exportSystem(particles, snapshotIteration, params, outputDir);
#if timeComponents
				//			hipEventRecord(stop{13],0);
				//			hipEventSynchronize(stop[13]);
				double fileTimeStop = omp_get_wtime();
				cudaTime[13] = (float)(fileTimeStop - fileTimeStart) * 1000; //file writing time in ms
				//printf("OMP thread number %u\n",omp_get_thread_num());
#endif
			}
#pragma omp section
			{
				/////////////////////////////////////////////////////////////////////////////////////////////////////
				//COMPUTE NEXT STRIDES WORTH OF ITERATIONS
				/////////////////////////////////////////////////////////////////////////////////////////////////////
				for (int ind2 = 0; ind2 < storageStride; ind2++)
				{
					TIME(0, calcHash << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););  //compute hash value
					TIME(1, thrust::sort_by_key(thrust::device_ptr<int>(pdParticlesHostMirror->gridParticleHash), thrust::device_ptr<int>(pdParticlesHostMirror->gridParticleHash + (*params).nTotal), thrust::device_ptr<int>(pdParticlesHostMirror->gridParticleIndex)););  //sort
					hipDeviceSynchronize();
					//dGridParticleHash is now sorted into ascending order
					//dGridParticleIndex is now sorted based on the rearrangement of dGridParticleHash


					/*  Debug
					hipDeviceSynchronize();
					printf("value of pdParticlesHostMirror->gridParticleHash%p\n",pdParticlesHostMirror->gridParticleHash);

					int* tempInt   = new int[nTotal];

					hipMemcpy(tempInt,pdParticlesHostMirror->gridParticleHash,nTotal*sizeof(int),hipMemcpyDeviceToHost);
					hipMemcpy(particles->x,pdParticlesHostMirror->x,nTotal*sizeof(double),hipMemcpyDeviceToHost);
					hipMemcpy(particles->y,pdParticlesHostMirror->y,nTotal*sizeof(double),hipMemcpyDeviceToHost);

					int cellMax = tempInt[0];
					for (int ind3 = 0;ind3<nTotal;ind3++)
					{if (tempInt[ind3]>cellMax) {cellMax = tempInt[ind3];}
					};
					printf("%d",cellMax);
					double k2 = 2;
					delete tempInt;
					printf("%d\n",ind2);
					*/

					//set all cells to empty
					hipMemset(pdParticlesHostMirror->cellStart, 0xffffffff, (*params).nCellsTotal * sizeof(int));
					hipDeviceSynchronize();
					//reorder data and find cell start
					TIME(2, reorderAndFindCellStart << <numberBlocks, threadsPerBlock, smemSize >> > (pdParticles, pdParams););
					/////////////////COMPUTE DV/DT////////////////////////////////////////////////////////////
					TIME(3, computedVdt << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););		//
					//TIME(4, computedRhodt << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					//////////////////////////////////////////////////////////////////////////////////////////


					////////////////MOVE PARTICLES, UPDATE VELOCITY, UPDATE DENISTY////////////////////
					TIME(4, updateVelWithXSPH << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					TIME(5, updateVelocity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					TIME(6, updateDensity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					//DEBUG(DEBUGdisplayDensity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););					   //
					/*
					if (iteration % 20 == 0)  //this is tied to "iteration" not the subiterator counting iterations per stride
					{
						TIME(7, reinitializeDensity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
						TIME(8, moveFilteredDensityToDensity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
						DEBUG(DEBUGdisplayMoveDensity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););				   //
					}
					*/
					TIME(7, copySortedRhoToDensity << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					TIME(8, updatePositionFreeParticles << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););  //have to re-bin in order to accurately reinitialize
					//TIME(11, computeShift << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					//TIME(12, computeShiftGrad << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					//TIME(12, computeDiv << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					//TIME(13, updateShift << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams););
					//move the boundary
					//we only need to cover the constrained particles
					if (numberBlocksConstrained > 0) {
						TIME(9, moveConstrainedParticlesD << <numberBlocks, threadsPerBlock >> > (pdParticles, pdParams, pdKinematicsFunction););
					}  //close constrained particles

		////////////////////////////////////////////////////////////////////////////////////////
		//HEART BEAT
		////////////////////////////////////////////////////////////////////////////////////////
					if (ind2 == (storageStride - 1))
					{
						hipEventRecord(tStop);
						hipEventSynchronize(tStop);
						hipEventElapsedTime(&elapsedTime, tStart, tStop);
						printf("%5.0d steps took %f ms; %f ms/step;OMP thread # % u;\n", storageStride, elapsedTime, elapsedTime / storageStride, omp_get_thread_num());
						//OMP thread # % u
						//omp_get_thread_num()
						hipEventRecord(tStart);

					}  //status update

					iteration++;  //increment global iteration count
					hipDeviceSynchronize();


				} // close iterating over a stride
			} // close omp section of iterating over a stride

//////////////////////////////////////////////////////////////////////////////////////////////////////////


		} //close OMP SECTIONS; IMPLICIT BARRIER FOR OMP THREADS


#if timeComponents

		double componentSum = 0;
		//compute elapsed times
		for (int ind3 = 0; ind3 < 13; ind3++) { //exclude file export
			componentSum += cudaTime[ind3];
		}

		printf("\n");
		for (int ind3 = 0; ind3 < 19; ind3++) {
			printf("Time for %s is %f ms, %f, %%\n", procedureNames[ind3], cudaTime[ind3], cudaTime[ind3] / componentSum * 100);
			cudaTime[ind3] = 0;
		}
		printf("componentSum                           is %f\n", componentSum);

#endif



	}  //check while loop to see if more iterations necessary

			//free the GPU memory
			//it is probably freed upon exit



	/*
			hipFree(dPos);
			hipFree(dmassRadius);
			hipFree(dpRho);
			hipFree(dVel);
			hipFree(dVelHalf);
			hipFree(dForce);
			hipFree(dpColor);
			hipFree(dpColorSorted);
			hipFree(dParams);

			hipFree(dSortedPos);
			hipFree(dSortedVel);

			hipFree(dsortedpRho);
			hipFree(dXSPHVel);
			hipFree(dGridParticleHash);
			hipFree(dGridParticleIndex);
			hipFree(dCellStart);
			hipFree(dCellEnd);
			hipFree(dInd1);

			*/

			//end time loop

	return;
}