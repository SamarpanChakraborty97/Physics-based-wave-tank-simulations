#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "stdio.h"
#include "smoothingKernels.cuh"

__global__ void moveFilteredDensityToDensity(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	double rhoRef = pparams->rRef;
	double tenVMaxSq = pparams->tenVMaxSq;
	if (index >= (*pparams).nTotal) return;

	//put filtered density back into "working" density 
	pparticles->sortedRho[index] = pparticles->sortedRhoFiltered[index];
	pparticles->sortedPressure[index] = computePressure(pparticles->sortedRho[index], rhoRef, tenVMaxSq);


	//	pparticles->density[index] = pparticles->unsortedRhoFiltered[index];


	return;
}