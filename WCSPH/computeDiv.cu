#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <iostream>
#include "smoothingKernels.cuh"
#include <stdio.h>

__device__ double divInCell2(int2 neighboor, int index, double posX, double posY, struct particleStructure* pparticles, struct paramsType* pparams);

__global__ void computeDiv(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//read particle data - host particle
	double posXi = pparticles->sortedX[index];  //these are sorted, I is the receiver
	double posYi = pparticles->sortedY[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double posDiv = 0; //position divergence

	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					posDiv += divInCell2(neighboor, index, posXi, posYi, pparticles, pparams);
				}
			}
		}
	}

	//pparticles->sortedShiftGradX = shiftGrad[0];
	//pparticles->sortedShiftGradY = shiftGrad[1];

	int originalIndex = pparticles->gridParticleIndex[index];
	pparticles->posDiv[originalIndex] = posDiv;


	return;
}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ double divInCell2(int2 neighboor, int index, double posXi, double posYi, struct particleStructure* pparticles, struct paramsType* pparams) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;
	//double constantSpikyImprovedD = pparams->spikyImprovedD;
	double constWendlandD = pparams->constwendlandD;

	int startIndex = pparticles->cellStart[hash];
	double div = 0;
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//remember to include self density
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			double rhoJ = pparticles->sortedRho[ind1]; // density of the neighbouring particle

			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dx = posXi - posXj;
			double dy = posYi - posYj;
			double rSq = dx * dx + dy * dy;
			double diffSq = 4*(*pparams).h2 - rSq;
			if (diffSq >= 0) {
				double r = sqrt(rSq);
				double rOh = r / pparams->h;
				double normalizedGradientInfluence = (1 / r) * wendlandD(constWendlandD, rOh);
				div += (m2 / rhoJ) * normalizedGradientInfluence * (dx * dx + dy * dy);

				//debug
				int index = blockIdx.x * blockDim.x + threadIdx.x;
				int origIndex = pparticles->gridParticleIndex[index];
				if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
					int senderIdx = pparticles->gridParticleIndex[ind1];
					printf("Ini D; it=%u, P# %u -> w/ %u at %f and %f \n", pparams->ind1, pparams->DEBUGpNum, senderIdx, posXj, posYj);
				}


			}; //end checking closeness
		};  //end the for loop
	};//end the if statement

	return div;
}