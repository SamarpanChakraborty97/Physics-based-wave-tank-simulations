#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "thrust/device_vector.h"


__global__ void reorderAndFindCellStart(struct particleStructure* pparticles, struct paramsType* pparams)
{
	extern __shared__ int sharedHash[];  //the size of this array is passed as the third arguement; it defaults to the blockDim.x; which won't work for us
	int index = blockIdx.x * blockDim.x + threadIdx.x;  //each thread treats a particle

	int hash;
	//# particle not multiple block size
	if (index < (*pparams).nTotal) {
		hash = pparticles->gridParticleHash[index]; //work on valid data; get 1D hash value

		sharedHash[threadIdx.x + 1] = hash;

		if (index > 0 && threadIdx.x == 0) {  //the first one in the block - get it from the previous block
			sharedHash[0] = pparticles->gridParticleHash[index - 1];
		}
	}

	__syncthreads();

	if (index < (*pparams).nTotal) {

		if (index == 0 || hash != sharedHash[threadIdx.x])  //only one thread/cell will satisfy this
		{
			pparticles->cellStart[hash] = index;  //store as a function of hash value
			if (index > 0) { pparticles->cellEnd[sharedHash[threadIdx.x]] = index; };  //rem sharedHash blockSize+1
		}

		if (index == (*pparams).nTotal - 1)
		{
			pparticles->cellEnd[hash] = index + 1;
		};

		int sortedIndex = pparticles->gridParticleIndex[index];  //this was sorted by thrust
		//double dt = pparams->dt;
		//double b = pparams->beta;
		//double relS = pparams->relaxStart;
		//double relE = pparams->relaxEnd;

		//cellStart and cellEnd contain the sorted particle indices which span i.e. begin and end 
		//each hash cell

		//not all hash cells will contain particles, so they are assigned a value of 0xffffffff
		//all particles will be associated with a cell.


		//sort arrays
		//sorted arrays are accessible by cellStart and cellEnd
		//for instance, assuming cell 5 is populated with a few particles
		//cellStart[5] & cellEnd[5] provide an indices which span those particles
		//the particles must be sorted so consecutive particles from cellStart[5] to cellEnd[5] are contained
		//within the cell


		//all individual properties of particles must be sorted
		//there may be a faster way to sort these arrays
		pparticles->sortedX[index] = pparticles->x[sortedIndex];
		pparticles->sortedY[index] = pparticles->y[sortedIndex];
		pparticles->sortedVx[index] = pparticles->vx[sortedIndex];
		pparticles->sortedVy[index] = pparticles->vy[sortedIndex];
		pparticles->sortedRho[index] = pparticles->density[sortedIndex];
		//pparticles->sortedPressure[index] = pparticles->pressure[sortedIndex];

		/*
		if ((pparticles->sortedX[index] > relS) && (pparticles->sortedX[index] < relE))
		{
			double dampV = b * dt * ((pparticles->sortedX[index] - relS) / (relE - pparticles->sortedX[index])) * ((pparticles->sortedX[index] - relS) / (relE - pparticles->sortedX[index]));
			pparticles->sortedVx[index] = pparticles->sortedVx[index] * (1 - dampV);
			pparticles->sortedVy[index] = pparticles->sortedVy[index] * (1 - dampV);
		}
		*/
		//if each particle had a unique smoothing length and mass, they would have to be sorted as sorted arrays as well.
		//pM,  pR
	};

	//printf("%d \n", pparticles->sortedX[index]);
//	if (index < 1){
//		printf("cell start[0] %d\n",dCellStart[0]);
//		printf("cell end[0] %d\n",dCellEnd[0]);
//	};




	return;
}