#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <iostream>
#include "smoothingKernels.cuh"
#include <stdio.h>

__device__ double shiftInCell2(int2 neighboor, int index, double posX, double posY, struct particleStructure* pparticles, struct paramsType* pparams);

__global__ void computeShift(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//read particle data - host particle
	double posXi = pparticles->sortedX[index];  //these are sorted, I is the receiver
	double posYi = pparticles->sortedY[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double shift = 0;	//need density
	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					shift += shiftInCell2(neighboor, index, posXi, posYi, pparticles, pparams);
				}
			}
		}
	}

	pparticles->sortedShift[index] = shift;   //this one is not used

//	printf("sorted Rho of a particle %u is %f\n",index, density);
	//int originalIndex = pparticles->gridParticleIndex[index];
	//pparticles->shift[originalIndex] = shift;   //this one is used


	//debug
	int origIndex = pparticles->gridParticleIndex[index];
	if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
		printf("Ini D; it=%u, Density of %u is %f \n", pparams->ind1, origIndex, shift);
	}


	//also initialize dRhodt = 0 for all particles; order doesn't matter
	//pparticles->sorteddRhodt[index] = 0;

	return;
}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ double shiftInCell2(int2 neighboor, int index, double posXi, double posYi, struct particleStructure* pparticles, struct paramsType* pparams) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	double shift = 0;
	double constWendland = pparams->constwendland;
	//double constPoly6 = pparams->constDensity;

	int startIndex = pparticles->cellStart[hash];
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//remember to include self density
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			double rhoJ = pparticles->sortedRho[ind1]; // density of the neighbouring particle
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double rSq = (posXi - posXj) * (posXi - posXj) + (posYi - posYj) * (posYi - posYj);
			//double diffSq = (*pparams).h2 - rSq;
			if ((rSq <= 4*(*pparams).h2) && (rSq > 0)) {
				double r = sqrt(rSq);
				double rOh = r / pparams->h;
				shift += (m2 / rhoJ) * wendland(constWendland, rOh); //   4/(pi h^2)*(1-r^2/h^2)^3

				//debug
				int index = blockIdx.x * blockDim.x + threadIdx.x;
				int origIndex = pparticles->gridParticleIndex[index];
				if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
					int senderIdx = pparticles->gridParticleIndex[ind1];
					printf("Ini D; it=%u, P# %u -> w/ %u at %f and %f \n", pparams->ind1, pparams->DEBUGpNum, senderIdx, posXj, posYj);
				}


			}; //end checking closeness
		};  //end the for loop
	};//end the if statement

	return shift;
}