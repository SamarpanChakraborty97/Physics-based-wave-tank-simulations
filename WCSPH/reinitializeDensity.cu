#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "stdio.h"
#include "smoothingKernels.cuh"


__device__ double2 computeComponents(int2 neighboor, int index, double posX, double posY, struct particleStructure* pparticles, struct paramsType* pparams);


__global__ void reinitializeDensity(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//read particle data - host particle
	double posXi = pparticles->sortedX[index];  //these are sorted
	double posYi = pparticles->sortedY[index];

	//get address in grid
	int tempX = floor((posXi - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posYi - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double2 numDenom = { 0,0 };	//need 
	double2 temp = { 0,0 };
	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					temp = computeComponents(neighboor, index, posXi, posYi, pparticles, pparams);
					numDenom.x += temp.x;
					numDenom.y += temp.y;
				}
			}
		}
	}




	//if particles exceede the boundaries they will have denom = 0 and rhoi = 0;
	//In this case, they are no longer an important part of the calculation, so 
	//set their rhoi = density of a single particle

	//rhoi = m2*(*pparams).constDensity*(1-rOhSq)*(1-rOhSq)*(1-rOhSq);
	// = m2*(*pparams).constDensity*(1-0)*(1-0)*(1-0);
	// = m2*(*pparams).constDensity;

	//debug
	int origIndex = pparticles->gridParticleIndex[index];
	if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
		printf("reini rho; it=%u, P# %u; num %f, denom %f \n", pparams->ind1, origIndex, numDenom.x, numDenom.y);
	}

	//double filteredDensity = numerator/denom;
	double filteredDensity = numDenom.x / numDenom.y;

	if (numDenom.y == 0)
	{
		//maybe keeping it the same is the answer
		filteredDensity = 2 * pparticles->mass[0] * (*pparams).constDensity;
		printf("particle exceeded boundary\n");
	}


	if (filteredDensity == 0) {
		printf("something is wrong\n");
	}


	//store the density in a temporary array
	pparticles->sortedRhoFiltered[index] = filteredDensity;  //sortedRhoFiltered no longer exists

	//write new filtered density back to original unsorted position
	//int originalIndex = pparticles->gridParticleIndex[index];
	//pparticles->unsortedRhoFiltered[originalIndex] = rhoi;


}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ double2 computeComponents(int2 neighboor, int index, double posXi, double posYi, struct particleStructure* pparticles, struct paramsType* pparams) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	int startIndex = pparticles->cellStart[hash];
	double2 numDenom = { 0,0 };
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {
			//no reason to include self in drhodt
			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles

			//The kernel is W = 
			double dxij = (posXi - posXj);
			double dyij = (posYi - posYj);

			double rSq = dxij * dxij + dyij * dyij;
			//double diffSq = 4*(*pparams).h2 - rSq;
			if ((rSq <= 4*(*pparams).h2) && (rSq >= 0))
			{
				double rhoj = pparticles->sortedRho[ind1];
				double dist = sqrt(rSq);
				double rOh = dist / pparams->h;

				double kernelInfluence = wendland(pparams->constwendland, rOh);
				numDenom.x += kernelInfluence * m2;        //numerator
				numDenom.y += kernelInfluence * m2 / rhoj;   //denominator




			};  //end the for loop
		};//end the if statement
	};

	return numDenom;
}