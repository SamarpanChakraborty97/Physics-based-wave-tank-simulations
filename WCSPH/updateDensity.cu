#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include "stdio.h"
#include "smoothingKernels.cuh"

__global__ void updateDensity(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	double rhoRef = pparams->rRef;
	double tenVMaxSq = pparams->tenVMaxSq;

	if (index < (*pparams).nTotal) {  //operate over all particles

		double dt = pparams->dt;

		//store the updated sorted density
		pparticles->sortedRho[index] += pparticles->sorteddRhodt[index] * dt;
		pparticles->sortedPressure[index] = computePressure(pparticles->sortedRho[index], rhoRef, tenVMaxSq);
		//printf("%d \n", pparticles->sortedRho[index]);

#if 0  //limit the density 
		if (> ) {
		}

#endif

	}

	return;
}