#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "SPH2DCPPCuda.h"
#include <stdio.h>

__device__ double2 XSPHVelocityInCell(int2 neighboor, int index, double posX, double posY, double velxi, double velyi, double rhoi, struct particleStructure* pparticles, struct paramsType* pparams);

__global__ void computeXSPHInfluence(struct particleStructure* pparticles, struct paramsType* pparams) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (*pparams).nTotal) return;

	//this is accessing the sorted velocities; but the unsorted veolicties have been updated with the new accelerations
	//it should access the unsorted velocities.


	double epsilon = (*pparams).epsilon;  //parameter for XSPH influence

	//read particle data - host particle
	double posX = pparticles->sortedX[index];  //these are sorted
	double posY = pparticles->sortedY[index];

	double velxi = pparticles->sortedVx[index];  //these are sorted
	double velyi = pparticles->sortedVy[index];

	double rhoi = pparticles->sortedRho[index];

	//get address in grid
	int tempX = floor((posX - (*pparams).globalOriginX) * (*pparams).cellSizeRecip);
	int tempY = floor((posY - (*pparams).globalOriginY) * (*pparams).cellSizeRecip);

	int2 gridPos = { tempX,tempY }; // grid position of host particle

	//examine neighbooring cells
	double2 XSPHVel = { 0,0 };	//need XSPH
	double2 temp = { 0,0 };
	for (int y = -1; y <= 1; y++) {
		int currentY = gridPos.y + y;
		if ((currentY > -1) && (currentY < (*pparams).nCellsY)) {

			for (int x = -1; x <= 1; x++) {
				int currentX = gridPos.x + x;
				if ((currentX > -1) && (currentX < (*pparams).nCellsX)) {
					int2 neighboor = { currentX,currentY };  //2D index in grid
					temp = XSPHVelocityInCell(neighboor, index, posX, posY, velxi, velyi, rhoi, pparticles, pparams);
					XSPHVel.x += temp.x;
					XSPHVel.y += temp.y;
				}
			}
		}
	}

	//write XSPH density back to original unsorted position
	int originalIndex = pparticles->gridParticleIndex[index];

	pparticles->XSPHVelX[originalIndex] = epsilon * XSPHVel.x;
	pparticles->XSPHVelY[originalIndex] = epsilon * XSPHVel.y;

	//DEBUG
	int origIndex = pparticles->gridParticleIndex[index];
	if ((origIndex == pparams->DEBUGpNum) && (pparams->DEBUGinfo == 1)) {
		printf("XSPHx %f, XSPHY %f \n", pparticles->XSPHVelX[origIndex], pparticles->XSPHVelY[origIndex]);
	}



	return;
}


// loop over the particles in the host cell and surrounding cells; compute density
//__device__ double densityInCell(int2 neighboor,int index,double posX,double posY,double2* dPosSorted,double2* massRadius,int* cellStart,int* cellEnd, struct paramsType* pparams) {
__device__ double2 XSPHVelocityInCell(int2 neighboor, int index, double posXi, double posYi, double velxi, double velyi, double rhoi, struct particleStructure* pparticles, struct paramsType* pparams) {

	//compute 1D hash value
	int hash = neighboor.y * (*pparams).nCellsX + neighboor.x;

	int startIndex = pparticles->cellStart[hash];
	double2 XSPHvel = { 0,0 };
	if (startIndex != 0xffffffff) {
		int endIndex = pparticles->cellEnd[hash];

		for (int ind1 = startIndex; ind1 < endIndex; ind1++) {

			double posXj = pparticles->sortedX[ind1];  //get position of sending particles
			double posYj = pparticles->sortedY[ind1];
			double m2 = pparticles->mass[0];  //mass; right now these are identical  for all particles
			//compute density;  We use Monaghan's formulation with Muller's skPoly6 smoothing kernel normalized to 2D
			//The kernel is W = 
			double dxji = (posXj - posXi);
			double dyji = (posYj - posYi);
			double rSq = dxji * dxji + dyji * dyji;
			double diffSq = (*pparams).h2 - rSq;
			if (diffSq > 0) {
				//don't include constrained particles
			//	int originalIndex = pparticles->gridParticleIndex[ind1];
			//  if (originalIndex<pparams->nFree) {
				if (1) {

					double h = pparams->h;
					double rhoj = pparticles->sortedRho[ind1];
					double vxj = pparticles->sortedVx[ind1];
					double vyj = pparticles->sortedVy[ind1];

					double dvxji = vxj - velxi;
					double dvyji = vyj - velyi;

					double rhoBarij = (rhoi + rhoj) / 2;
					double rOh = sqrt(rSq) / h;
					double rOhSq = rOh * rOh;
					double mutualInfluence = m2 / rhoBarij * pparams->constDensity * (1 - rOhSq) * (1 - rOhSq) * (1 - rOhSq);

					XSPHvel.x += mutualInfluence * dvxji;
					XSPHvel.y += mutualInfluence * dvyji;

				}; //end excluding constrained particles
			}; //end checking closeness
		};  //end the for loop
	};//end the if statement - populated cells

	return XSPHvel;
}